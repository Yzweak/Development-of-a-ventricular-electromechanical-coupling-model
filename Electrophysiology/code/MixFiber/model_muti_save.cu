#include "hip/hip_runtime.h"
//
// Created by z on 23-4-19.
//
#include "model.cuh"
#include <iostream>
#include <vector>
#include <fstream>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>

using namespace std;

struct Atria_2D {
    double(*V)[YD];
    double(*Cai)[YD];
    double(*CaSR)[YD];
    double(*CaSS)[YD];
    double(*Nai)[YD];
    double(*Ki)[YD];
    double(*M)[YD];
    double(*H)[YD];
    double(*J)[YD];
    double(*Xr1)[YD];
    double(*Xr2)[YD];
    double(*Xs)[YD];
    double(*Rr)[YD];
    double(*S)[YD];
    double(*D)[YD];
    double(*Ff)[YD];
    double(*F2)[YD];
    double(*FCass)[YD];
    double(*RR)[YD];
    double(*OO)[YD];
    double(*Itot)[YD];
    double(*du)[YD];
    cell_type(*type)[YD];
    cell_location(*location)[YD];
    int(*state)[YD];

    double(*dCai)[YD];
    double(*IbCa)[YD];
    double(*IpCa)[YD];
    double(*INaCa)[YD];
    double(*Iup)[YD];
    double(*Ileak)[YD];
    double(*Ixfer)[YD];

};


void size_array(Atria_2D *(&A), int len)
{
    A = new Atria_2D[len];
}


int* splitStr(char* s)
{
    int* res = new int[4];
    int j = 0;
    for(int i=0;i<4;i++)
    {
        string ss = "";
        while(s[j] != ',' && s[j] != '\0')
        {
            ss += s[j];
            j++;
        }
        res[i] = atoi(ss.c_str());
        if(s[j] == '\r')
            break;
        j++;
    }
    return res;
}

void readMatrixFile(int** matrix)
{
    ifstream in;
    in.open(R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\ventricle_file\matrix.txt)",ios::in);
    if(!in){
        printf("File read Failed!\n");
        return;
    }
    char s[30];
    int i = 0;
    while(in.getline(s,sizeof(s)))
    {
        int* res = splitStr(s);
        for(int j=0;j<4;j++)
            matrix[i][j] = res[j];
        i++;
    }
    in.close();
}

void readFile(int* array, int mode)
{
    ifstream in;
    if(mode == 1)
        in.open(R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\ventricle_file\order.txt)",ios::in);
    else if(mode == 2)
        in.open(R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\ventricle_file\ischemia-1a.txt)",ios::in);
    else if(mode == 3)
        in.open(R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\ventricle_file\ischemia-1b.txt)",ios::in);
    else if(mode == 4)
        in.open(R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\ventricle_file\MI-short.txt)",ios::in);
    else if(mode == 5)
        in.open(R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\ventricle_file\MI-long.txt)",ios::in);
    if(!in){
        printf("File read Failed!\n");
        return;
    }
    char s[10];
    int i = 0;
    while(in.getline(s,sizeof(s)))
    {
        int j = 0;
        string ss = "";
        while (s[j] != '\0')
        {
            ss += s[j];
            j++;
        }
        array[i] = atoi(ss.c_str());
        i++;
    }
    in.close();
}

void InitCanine(Atria_2D* , const int*, const int*, const int*, const int*, const int*);
void freeCanine(Atria_2D*);
void cudaInitCanine(Atria_2D*);
void memcpy_hostToDev_Canine(Atria_2D* (&), Atria_2D* (&));
__global__ void kernel_RunCell(Atria_2D*, double);
__device__ void get_Itot(Atria_2D*, int, int, double);
void get_du(Atria_2D*,int**);
void memcpy_devToHost_Canine(Atria_2D* (&), Atria_2D* (&));
void cudaFreeCanine(Atria_2D* );
void WriteBackFile(const double* t,Atria_2D* c0);

/******************************************************************/

int main(int argc, char **argv)
{

    hipError_t err = hipSuccess;
    clock_t begin, finish;

    cout << endl <<  "Start simulation!" << endl << endl;
    /*****   near_matrix   ************************************************/
    int** matrix = new int*[rows];
    for(int i=0;i<rows;i++)
    {
        int* matrix_line = new int[cols];
        matrix[i] = matrix_line;
    }
    int* order = new int[stimNum];
    int* state_1a = new int[ischemia_1a_num];
    int* state_1b = new int[ischemia_1b_num];
    int* state_short = new int[MI_short_num];
    int* state_long = new int[MI_long_num];
    readMatrixFile(matrix);
    readFile(order,1);
    readFile(state_1a,2);
    readFile(state_1b,3);
    readFile(state_short,4);
    readFile(state_long,5);

    /*****   host   ************************************************/
    Atria_2D *cc;
    cc = new Atria_2D[1];
    InitCanine(cc,order,state_1a,state_1b,state_short,state_long);

    /*****  用于过渡   **********************************************/
    Atria_2D *cc_temp;
    size_array(cc_temp, 1);
    cudaInitCanine(cc_temp);

    /*****  device   ************************************************/
    Atria_2D *cc_dev;
    hipMalloc((void**)&cc_dev, 1 * sizeof(Atria_2D));


    /***  将host端数据复制到device端  *********************************/
    memcpy_hostToDev_Canine(cc_temp, cc);
    err = hipMemcpy(cc_dev, cc_temp, 1 * sizeof(Atria_2D), hipMemcpyHostToDevice);

    /*****  threads and blocks  ***************************************/
    dim3 threadPerBlock(8, 8);
    dim3 blockPerGrid(8, 8);


    /**  pacing     **************************************************/
    double period = 1000;
    double tbegin = 100;
    double tend = tbegin + stimduration;
    double STOPTIME = 100000;   //100000;
    double time = 0;
    double Istim = 0;

    begin = clock();

    for (int step = 0; time <= STOPTIME; step++)
    {
        if (time >= tbegin && time <= tend)
            Istim = stimstrength;
        if (time > tend)
        {
            Istim = 0.;
            tbegin = tbegin + period;
            tend = tbegin + stimduration;
        }
        if(time >= 99000 && step % 50 == 0)
        {
            err = hipMemcpy(cc->V, cc_temp->V, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
            err = hipMemcpy(cc->Cai, cc_temp->Cai, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
            err = hipMemcpy(cc->dCai, cc_temp->dCai, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
            err = hipMemcpy(cc->IbCa, cc_temp->IbCa, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
            err = hipMemcpy(cc->IpCa, cc_temp->IpCa, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
            err = hipMemcpy(cc->INaCa, cc_temp->INaCa, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
            err = hipMemcpy(cc->Iup, cc_temp->Iup, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
            err = hipMemcpy(cc->Ileak, cc_temp->Ileak, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
            err = hipMemcpy(cc->Ixfer, cc_temp->Ixfer, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
            WriteBackFile(&time,cc);
        }
        kernel_RunCell <<<blockPerGrid, threadPerBlock >>>(cc_dev, Istim);
        hipDeviceSynchronize();

        err = hipMemcpy(cc->V, cc_temp->V, XD*YD*sizeof(double), hipMemcpyDeviceToHost); //GPU to CPU
        get_du(cc,matrix);
        err = hipMemcpy(cc_temp->V, cc->V, XD*YD*sizeof(double), hipMemcpyHostToDevice);
        time += HT;
    }


    finish = clock();
    double time1 = (double)(finish - begin) / CLOCKS_PER_SEC;
    std::cout << "all simulation time (min) =" << time1 / 60 << endl << endl;

    /****** FREE-host ********/
    freeCanine(cc);
    delete[] cc;
    delete[] matrix;
    delete[] order;
    /****** FREE-device ********/
    cudaFreeCanine(cc_temp);
    delete[] cc_temp;
    hipFree(cc_dev);
    return 0;

}



void WriteBackFile(const double* t,Atria_2D* c0)
{
    char name1[300];
    char name2[300];
    char name3[300];
    char name4[300];
    char name5[300];
    char name6[300];
    char name7[300];
    char name8[300];
    char name9[300];

    sprintf(name1,"%s",R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\output\calcium.dat)");
    sprintf(name2,"%s",R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\output\volt.dat)");
    sprintf(name3,"%s",R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\output\dCai.dat)");
    sprintf(name4,"%s",R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\output\IbCa.dat)");
    sprintf(name5,"%s",R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\output\IpCa.dat)");
    sprintf(name6,"%s",R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\output\INaCa.dat)");
    sprintf(name7,"%s",R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\output\Iup.dat)");
    sprintf(name8,"%s",R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\output\Ileak.dat)");
    sprintf(name9,"%s",R"(C:\Users\dell\Desktop\CLionProjects\MixFiber\output\Ixfer.dat)");
    ofstream oo1(name1,ios::app);
    ofstream oo2(name2,ios::app);
    ofstream oo3(name3,ios::app);
    ofstream oo4(name4,ios::app);
    ofstream oo5(name5,ios::app);
    ofstream oo6(name6,ios::app);
    ofstream oo7(name7,ios::app);
    ofstream oo8(name8,ios::app);
    ofstream oo9(name9,ios::app);
    if(!oo1){
        printf("cannot open file %s\n",name1);
        exit(1);
    }
    if(!oo2){
        printf("cannot open file %s\n",name2);
        exit(1);
    }

    oo1 << floor(*t-99000) << "\t";
    oo2 << floor(*t-99000) << "\t";
    oo3 << floor(*t-99000) << "\t";
    oo4 << floor(*t-99000) << "\t";
    oo5 << floor(*t-99000) << "\t";
    oo6 << floor(*t-99000) << "\t";
    oo7 << floor(*t-99000) << "\t";
    oo8 << floor(*t-99000) << "\t";
    oo9 << floor(*t-99000) << "\t";
    for (int j = 0;j<YD;j++)
    {
        for(int i=0;i<XD;i++)
        {
            if(j * YD + i < cellNum)
            {
                oo1 << c0->Cai[i][j] << "\t";
                oo2 << c0->V[i][j] << "\t";
                oo3 << c0->dCai[i][j] << "\t";
                oo4 << c0->IbCa[i][j] << "\t";
                oo5 << c0->IpCa[i][j] << "\t";
                oo6 << c0->INaCa[i][j] << "\t";
                oo7 << c0->Iup[i][j] << "\t";
                oo8 << c0->Ileak[i][j] << "\t";
                oo9 << c0->Ixfer[i][j] << "\t";
            }
            else
            {
                oo1 <<"\n";
                oo2 <<"\n";
                oo3 <<"\n";
                oo4 <<"\n";
                oo5 <<"\n";
                oo6 <<"\n";
                oo7 <<"\n";
                oo8 <<"\n";
                oo9 <<"\n";

                oo1.close();
                oo2.close();
                oo3.close();
                oo4.close();
                oo5.close();
                oo6.close();
                oo7.close();
                oo8.close();
                oo9.close();
                return;
            }

        }
    }

}

void InitCanine(Atria_2D* c0, const int* order, const int* state_1a, const int* state_1b, const int* state_short, const int* state_long)
{
    c0->V = new double[XD][YD];
    c0->Cai = new double[XD][YD];
    c0->CaSR = new double[XD][YD];
    c0->CaSS = new double[XD][YD];
    c0->Nai = new double[XD][YD];
    c0->Ki = new double[XD][YD];
    c0->M = new double[XD][YD];
    c0->H = new double[XD][YD];
    c0->J = new double[XD][YD];
    c0->Xr1 = new double[XD][YD];
    c0->Xr2 = new double[XD][YD];
    c0->Xs = new double[XD][YD];
    c0->Rr = new double[XD][YD];
    c0->S = new double[XD][YD];
    c0->D = new double[XD][YD];
    c0->Ff = new double[XD][YD];
    c0->F2 = new double[XD][YD];
    c0->FCass = new double[XD][YD];
    c0->RR = new double[XD][YD];
    c0->OO = new double[XD][YD];
    c0->Itot = new double[XD][YD];
    c0->du = new double[XD][YD];
    c0->type = new cell_type[XD][YD];
    c0->location = new cell_location[XD][YD];
    c0->state = new int[XD][YD];

    c0->dCai = new double[XD][YD];
    c0->IbCa = new double[XD][YD];
    c0->IpCa = new double[XD][YD];
    c0->INaCa = new double[XD][YD];
    c0->Iup = new double[XD][YD];
    c0->Ileak = new double[XD][YD];
    c0->Ixfer = new double[XD][YD];

    for (int y = 0; y < YD; y++) {
        for (int x = 0; x < XD; x++) {

            int temp = y * YD + x;

            if(temp >= cellNum)
            {
                c0->type[x][y] = Nothing;
                c0->location[x][y] = EPI;
                c0->state[x][y] = 0;
            }
            else
            {
                c0->state[x][y] = 0;
                c0->location[x][y] = EPI;
                c0->type[x][y] = PM;
                for (int i=0;i<100;i++)
                {
                    if(temp == order[i])
                    {
                        c0->type[x][y] = SAN;
                        break;
                    }
                }
                bool is_ischemia_1a = false;
                bool is_ischemia_1b = false;
                bool is_MI_short = false;

                for (int i=0;i<ischemia_1a_num;i++)
                {
                    if(temp == state_1a[i])
                    {
                        c0->state[x][y] = 1;
                        is_ischemia_1a = true;
                        break;
                    }
                }
                if(is_ischemia_1a) continue;
                for (int i=0;i<ischemia_1b_num;i++)
                {
                    if(temp == state_1b[i])
                    {
                        c0->state[x][y] = 2;
                        is_ischemia_1b = true;
                        break;
                    }
                }
                if(is_ischemia_1b) continue;
                for (int i=0;i<MI_short_num;i++)
                {
                    if(temp == state_short[i])
                    {
                        c0->state[x][y] = 3;
                        is_MI_short = true;
                        break;
                    }
                }
                if(is_MI_short) continue;
                for (int i=0;i<MI_long_num;i++)
                {
                    if(temp == state_long[i])
                    {
                        c0->state[x][y] = 4;
                        break;
                    }
                }
            }
        }
    }

    for (int y = 0; y < YD; y++)
    {
        for (int x = 0; x < XD; x++)
        {
            //assign cell state
            if(c0->state[x][y] == 0 || c0->state[x][y] == 3 || c0->state[x][y] == 4)
            {
                c0->V[x][y] = -86.2;
                c0->Cai[x][y] = 0.00007;
                c0->CaSR[x][y] = 1.3;
                c0->CaSS[x][y] = 0.00007;
                c0->Nai[x][y] = 7.67;
                c0->Ki[x][y] = 138.3;
                c0->M[x][y] = 0.0;
                c0->H[x][y] = 0.75;
                c0->J[x][y] = 0.75;
                c0->Xr1[x][y] = 0.0;
                c0->Xr2[x][y] = 1.0;
                c0->Xs[x][y] = 0.0;
                c0->Rr[x][y] = 0.0;
                c0->S[x][y] = 1.0;
                c0->D[x][y] = 0.0;
                c0->Ff[x][y] = 1.0;
                c0->F2[x][y] = 1.0;
                c0->FCass[x][y] = 1.0;
                c0->RR[x][y] = 1.0;
                c0->OO[x][y] = 0.0;
                c0->Itot[x][y] = 0.0;
                c0->du[x][y] = 0.0;
            }
            else if(c0->state[x][y] == 1)
            {
                c0->V[x][y] = -70.342786;
                c0->Cai[x][y] = 0.000095738714626814048;
                c0->CaSR[x][y] = 2.874115913016119800000;
                c0->CaSS[x][y] = 0.000302877364231207700;
                c0->Nai[x][y] = 6.525357256767184900000;
                c0->Ki[x][y] = 141.870292899352390000000;
                c0->M[x][y] = 0.033655635139547148000;
                c0->H[x][y] = 0.211053852569920620000;
                c0->J[x][y] = 0.180147754598375870000;
                c0->Xr1[x][y] = 0.024098542643841897000;
                c0->Xr2[x][y] = 0.323941263226489200000;
                c0->Xs[x][y] = 0.009339544433470185100;
                c0->Rr[x][y] = 0.000000288926510767676;
                c0->S[x][y] = 0.999957603371733010000;
                c0->D[x][y] = 0.000164510100997893920;
                c0->Ff[x][y] = 0.980144782112325140000;
                c0->F2[x][y] = 0.995728345214580090000;
                c0->FCass[x][y] = 0.999969154239247040000;
                c0->RR[x][y] = 0.989633139742973960000;
                c0->OO[x][y] = 0.000000171795851160072;
                c0->Itot[x][y] = 0.0;
                c0->du[x][y] = 0.0;
            }
            else if(c0->state[x][y] == 2)
            {
                c0->V[x][y] = -69.756526;
                c0->Cai[x][y] = 0.000110191555337951770;
                c0->CaSR[x][y] = 2.899895833813058000000;
                c0->CaSS[x][y] = 0.000261707275030859110;
                c0->Nai[x][y] = 8.443153663718010900000;
                c0->Ki[x][y] = 139.046086118217600000000;
                c0->M[x][y] = 0.037396266233898176000;
                c0->H[x][y] = 0.193531112266500660000;
                c0->J[x][y] = 0.170193474094604810000;
                c0->Xr1[x][y] = 0.013720983480392804000;
                c0->Xr2[x][y] = 0.318614799111897550000;
                c0->Xs[x][y] = 0.009711833626984207200;
                c0->Rr[x][y] = 0.000000318575104674730;
                c0->S[x][y] = 0.999952334011770310000;
                c0->D[x][y] = 0.000177883400051902350;
                c0->Ff[x][y] = 0.988991405078883500000;
                c0->F2[x][y] = 0.995358028564252040000;
                c0->FCass[x][y] = 0.999978028985132990000;
                c0->RR[x][y] = 0.992241352890124500000;
                c0->OO[x][y] = 0.000000129042183903380;
                c0->Itot[x][y] = 0.0;
                c0->du[x][y] = 0.0;
            }
        }
    }
}

void cudaInitCanine(Atria_2D* c0)
{
    hipError_t err = hipMalloc((void **)&c0->V, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Cai, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->CaSR, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->CaSS, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Nai, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Ki, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->M, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->H, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->J, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Xr1, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Xr2, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Xs, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Rr, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->S, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->D, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Ff, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->F2, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->FCass, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->RR, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->OO, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Itot, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->du, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->type, sizeof(cell_type)* XD*YD);
    err = hipMalloc((void **)&c0->location, sizeof(cell_location)* XD*YD);
    err = hipMalloc((void **)&c0->state, sizeof(int)* XD*YD);

    err = hipMalloc((void **)&c0->dCai, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->IbCa, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->IpCa, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->INaCa, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Iup, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Ileak, sizeof(double)* XD*YD);
    err = hipMalloc((void **)&c0->Ixfer, sizeof(double)* XD*YD);

    if (err != hipSuccess)
    {
        std::cerr << err << endl;
        // fprintf(stderr, "Failed to allocate device in memcpy_hostToDev_Canine\n", hipGetErrorString(err));
        printf("Failed to allocate device in cudaInitCanine\n");
        exit(EXIT_FAILURE);
    }
}

void memcpy_hostToDev_Canine(Atria_2D* (&c0_dev), Atria_2D* (&c0))
{
    hipError_t err = hipMemcpy(c0_dev->V, c0->V, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Cai, c0->Cai, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->CaSR, c0->CaSR, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->CaSS, c0->CaSS, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Nai, c0->Nai, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Ki, c0->Ki, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->M, c0->M, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->H, c0->H, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->J, c0->J, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Xr1, c0->Xr1, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Xr2, c0->Xr2, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Xs, c0->Xs, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Rr, c0->Rr, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->S, c0->S, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->D, c0->D, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Ff, c0->Ff, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->F2, c0->F2, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->FCass, c0->FCass, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->RR, c0->RR, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->OO, c0->OO, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Itot, c0->Itot, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->du, c0->du, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->type, c0->type, XD*YD*sizeof(cell_type), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->location, c0->location, XD*YD*sizeof(cell_location), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->state, c0->state, XD*YD*sizeof(int), hipMemcpyHostToDevice);


    err = hipMemcpy(c0_dev->dCai, c0->dCai, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->IbCa, c0->IbCa, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->IpCa, c0->IpCa, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->INaCa, c0->INaCa, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Iup, c0->Iup, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Ileak, c0->Ileak, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(c0_dev->Ixfer, c0->Ixfer, XD*YD*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        std::cerr << err << endl;
        // fprintf(stderr, "Failed to allocate device in memcpy_hostToDev_Canine\n", hipGetErrorString(err));
        printf("Failed to allocate device in memcpy_hostToDev_Canine\n");
        exit(EXIT_FAILURE);
    }
}

void memcpy_devToHost_Canine(Atria_2D* (&c0), Atria_2D* (&c0_dev))
{
    hipError_t err = hipMemcpy(c0->V, c0_dev->V, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Cai, c0_dev->Cai, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->CaSR, c0_dev->CaSR, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->CaSS, c0_dev->CaSS, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Nai, c0_dev->Nai, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Ki, c0_dev->Ki, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->M, c0_dev->M, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->H, c0_dev->H, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->J, c0_dev->J, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Xr1, c0_dev->Xr1, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Xr2, c0_dev->Xr2, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Xs, c0_dev->Xs, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Rr, c0_dev->Rr, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->S, c0_dev->S, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->D, c0_dev->D, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Ff, c0_dev->Ff, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->F2, c0_dev->F2, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->FCass, c0_dev->FCass, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->RR, c0_dev->RR, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->OO, c0_dev->OO, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Itot, c0_dev->Itot, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->du, c0_dev->du, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->type, c0_dev->type, XD*YD*sizeof(cell_type), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->location, c0_dev->location, XD*YD*sizeof(cell_location), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->state, c0_dev->state, XD*YD*sizeof(int), hipMemcpyDeviceToHost);

    err = hipMemcpy(c0->dCai, c0_dev->dCai, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->IbCa, c0_dev->IbCa, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->IpCa, c0_dev->IpCa, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->INaCa, c0_dev->INaCa, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Iup, c0_dev->Iup, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Ileak, c0_dev->Ileak, XD*YD*sizeof(double), hipMemcpyDeviceToHost);
    err = hipMemcpy(c0->Ixfer, c0_dev->Ixfer, XD*YD*sizeof(double), hipMemcpyDeviceToHost);


}

void get_du(Atria_2D* c0,int** matrix)
{
    for (int j=0;j<YD;j++)
    {
        for (int i=0;i<XD;i++)
        {
            int index = j * YD + i;
            if(index < cellNum)
            {
                int index1 = matrix[index][0];
                int index2 = matrix[index][1];
                int index3 = matrix[index][2];
                int index4 = matrix[index][3];
                if(index3 == -1)
                    c0->du[i][j] = Dlong * (c0->V[index1%YD][index1/YD] + c0->V[index2%YD][index2/YD] - 2*c0->V[index%YD][index/YD]) /  (dx*dx);
                else
                {
                    if(index4 == -1)
                        c0->du[i][j] = Dlong * (c0->V[index1%YD][index1/YD] + c0->V[index2%YD][index2/YD] + c0->V[index3%YD][index3/YD] - 3*c0->V[index%YD][index/YD]) /  (dx*dx);
                    else
                        c0->du[i][j] = Dlong * (c0->V[index1%YD][index1/YD] + c0->V[index2%YD][index2/YD] + c0->V[index3%YD][index3/YD] + c0->V[index4%YD][index4/YD] - 4*c0->V[index%YD][index/YD]) / (dx*dx);
                }
                if(c0->state[i][j] == 2) c0->du[i][j] *= 0.5;
                c0->V[i][j] =  c0->V[i][j] + HT * c0->du[i][j];
            }
            else
            {
                c0->du[i][j] = 0;
            }
        }
    }
}

__global__ void kernel_RunCell(Atria_2D* c0, double Istim)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (j * YD + i >= cellNum) //越界保护
        return;

    /****** Voltage update ********/
    get_Itot(c0, i, j, Istim);//已在刺激区域加入Istim
    c0->V[i][j] = c0->V[i][j] - HT * c0->Itot[i][j];
}

__device__ void get_Itot(Atria_2D* s0, int i, int j, double Istim)
{
    double sm = s0->M[i][j];
    double sh = s0->H[i][j];
    double sj = s0->J[i][j];
    double sxr1 = s0->Xr1[i][j];
    double sxr2 = s0->Xr2[i][j];
    double sxs = s0->Xs[i][j];
    double ss = s0->S[i][j];
    double sr = s0->Rr[i][j];
    double sd = s0->D[i][j];
    double sf = s0->Ff[i][j];
    double sf2 = s0->F2[i][j];
    double sfcass = s0->FCass[i][j];
    double sRR = s0->RR[i][j];
    double sOO = s0->OO[i][j];
    double svolt = s0->V[i][j];
    double Cai = s0->Cai[i][j];
    double CaSR = s0->CaSR[i][j];
    double CaSS = s0->CaSS[i][j];
    double Nai = s0->Nai[i][j];
    double Ki = s0->Ki[i][j];
    cell_type type = s0->type[i][j];
    cell_location location = s0->location[i][j];
    int state = s0->state[i][j];

    double Ko = 0;
    if (state == 0 || state == 4)
        Ko = 5.4;
    else
        Ko = 8.0;
    //Needed to compute currents
    double Ek = RTONF * (log((Ko / Ki)));
    double Ena = RTONF * (log((Nao / Nai)));
    double Eks = RTONF * (log((Ko + pKNa * Nao) / (Ki + pKNa * Nai)));
    double Eca = 0.5 * RTONF * (log((Cao / Cai)));
    double Ak1 = 0.1 / (1. + exp(0.06 * (svolt - Ek - 200)));
    double Bk1 = (3. * exp(0.0002 * (svolt - Ek + 100)) + exp(0.1 * (svolt - Ek - 10))) / (1. + exp(-0.5 * (svolt - Ek)));
    double rec_iK1 = Ak1 / (Ak1 + Bk1);
    double rec_iNaK = (1. / (1. + 0.1245 * exp(-0.1 * svolt * F / (R * T)) + 0.0353 * exp(-svolt * F / (R * T))));
    double rec_ipK = 1. / (1. + exp((25 - svolt) / 5.98));
    //Compute currents depend on cell states
    double cof_INa=1.0, cof_ICaL=1.0, cof_Ito=1.0, cof_IKr=1.0, cof_IKs=1.0, cof_INaCa=1.0, cof_INaK=1.0, cof_IbCa=1.0;
    switch (state)
    {
        case 1: // Ischemia-1a
            cof_INa = 0.887;
            cof_ICaL = 0.8;
            cof_Ito = 0.5;
            cof_IKs = 0.781;
            break;
        case 2: // Ischemia-1b
            cof_ICaL = 0.5;
            cof_INaCa = 0.6;
            cof_INaK = 0.46;
            cof_IbCa = 1.3;
            break;
        case 3: // MI-short
            cof_INa = 0.38;
            cof_ICaL = 0.62;
            cof_Ito = 0.37;
            cof_IKr = 0.30;
            cof_IKs = 0.20;
            break;
        case 4: // MI-long
            cof_INa = 0.38;
            cof_ICaL = 0.31;
            cof_Ito = 0.5;
            cof_IKr = 0.30;
            cof_IKs = 0.20;
            break;
        default:
            break;
    }
    double Gks,Gto;
    if (location == EPI)
    {
        Gks = 0.392;
        Gto = 0.294;
    }
    else if (location == ENDO)
    {
        Gks = 0.392;
        Gto = 0.073;
    }
    else
    {
        Gks = 0.098;
        Gto = 0.294;
    }
    double INa = cof_INa * GNa * sm * sm * sm * sh * sj * (svolt - Ena);//Zhai change
    double ICaL = cof_ICaL * GCaL * sd * sf * sf2 * sfcass * 4 * (svolt - 15) * (F * F / (R * T)) * (0.25 * exp(2 * (svolt - 15) * F / (R * T)) * CaSS - Cao) / (exp(2 * (svolt - 15) *             F / (R * T)) - 1.);
    double Ito = cof_Ito * Gto * sr * ss * (svolt - Ek);
    double IKr = cof_IKr * Gkr * sqrt(Ko / 5.4) * sxr1 * sxr2 * (svolt - Ek);
    double IKs = cof_IKs * Gks * sxs * sxs * (svolt - Eks);
    double IK1 = GK1 * rec_iK1 * (svolt - Ek);
    double INaCa = cof_INaCa * knaca * (1. / (KmNai * KmNai * KmNai + Nao * Nao * Nao)) * (1. / (KmCa + Cao)) * (1. / (1 + ksat * exp((n - 1) * svolt * F / (R * T)))) *
                   (exp(n * svolt * F / (R * T)) * Nai * Nai * Nai * Cao - exp((n - 1) * svolt * F / (R * T)) * Nao * Nao * Nao * Cai * 2.5);
    double INaK = cof_INaK * knak * (Ko / (Ko + KmK)) * (Nai / (Nai + KmNa)) * rec_iNaK;
    double IpCa = GpCa * Cai / (KpCa + Cai);
    double IpK = GpK * rec_ipK * (svolt - Ek);
    double IbNa = GbNa * (svolt - Ena);
    double IbCa = cof_IbCa * GbCa * (svolt - Eca);

    // 补充IKATP电流 根据：2011_Experiment-model interaction for analysis of epicardial activation during human ventricular fibrillation with global myocardial ischaemia
    double GKATP = 3.9;
    double H = 2.0;
    double nn = 0.24;
    double ATPi = 4.6;
    double Khalf = 0.25;
    double IKATP = GKATP * (1 / (1 + pow(ATPi / Khalf, H))) * pow(Ko / 5.4, nn) * (svolt - Ek);
    //将ORD模型里的INaL加入初始化INaL用到的参数
    double mL = 0;
    double hL = 1;
    double mLss = 1.0 / (1.0 + exp((-(svolt + 42.85)) / 5.264));
    double tmL = 1.0 / (6.765 * exp((svolt + 11.64) / 34.77) + 8.552 * exp(-(svolt + 77.42) / 5.955));
    mL = mLss - (mLss - mL) * exp(-HT / tmL);
    double hLss = 1.0 / (1.0 + exp((svolt + 87.61) / 7.488));
    double thL = 200.0;
    hL = hLss - (hLss - hL) * exp(-HT / thL);
    double cof_NaL = state == 0 ? 1.5 : 1;
    double INaL = cof_NaL * GNaL * (svolt - Ena) * mL * hL;

    switch (state)
    {
        case 0: // Normal-Cell
            IKATP = 0.0;
            INaL = 0.0;
            break;
        case 1: // Ischemia-1a
            INaL *= 1.5;
            break;
        case 2: // Ischemia-1b
            break;
        case 3: // MI-short
            break;
        case 4: // MI-long
            IKATP = 0.0;
            INaL = 0.0;
            break;
        default:
            break;
    }

    if(type == SAN)
        s0->Itot[i][j] = IKr + IKs + IK1 + Ito + INa + IbNa + ICaL + IbCa + INaK + INaCa + IpCa + IpK + IKATP + INaL + Istim;
    else
        s0->Itot[i][j] = IKr + IKs + IK1 + Ito + INa + IbNa + ICaL + IbCa + INaK + INaCa + IpCa + IpK + IKATP + INaL;
    //update concentrations
    double kCaSR = maxsr - ((maxsr - minsr) / (1 + (EC / CaSR) * (EC / CaSR)));
    double k1 = k1_ / kCaSR;
    double k2 = k2_ * kCaSR;
    double dRR = k4 * (1 - sRR) - k2 * CaSS * sRR;
    s0->RR[i][j] += HT * dRR;
    s0->OO[i][j] = k1 * CaSS * CaSS * sRR / (k3 + k1 * CaSS * CaSS);


    double Irel = Vrel * sOO * (CaSR - CaSS);
    double Ileak = Vleak * (CaSR - Cai);
    double Iup = Vmaxup / (1. + ((Kup * Kup) / (Cai * Cai)));
    double Ixfer = Vxfer * (CaSS - Cai);


    double CaCSQN = Bufsr * CaSR / (CaSR + Kbufsr);
    double dCaSR = HT * (Iup - Irel - Ileak);
    double bjsr = Bufsr - CaCSQN - dCaSR - CaSR + Kbufsr;
    double cjsr = Kbufsr * (CaCSQN + dCaSR + CaSR);
    s0->CaSR[i][j] = (sqrt(bjsr * bjsr + 4 * cjsr) - bjsr) / 2;

    double inverseVcF2 = 1 / (2 * Vc * F);
    double inverseVcF = 1. / (Vc * F);
    double inversevssF2 = 1 / (2 * Vss * F);
    double CaSSBuf = Bufss * CaSS / (CaSS + Kbufss);
    double dCaSS = HT * (-Ixfer * (Vc / Vss) + Irel * (Vsr / Vss) + (-ICaL * inversevssF2 * CAPACITANCE));
    double bcss = Bufss - CaSSBuf - dCaSS - CaSS + Kbufss;
    double ccss = Kbufss * (CaSSBuf + dCaSS + CaSS);
    s0->CaSS[i][j] = (sqrt(bcss * bcss + 4 * ccss) - bcss) / 2;


    double CaBuf = Bufc * Cai / (Cai + Kbufc);
    double dCai = HT * ((-(IbCa + IpCa - 2 * INaCa) * inverseVcF2 * CAPACITANCE) - (Iup - Ileak) * (Vsr / Vc) + Ixfer);
    double bc = Bufc - CaBuf - dCai - Cai + Kbufc;
    double cc = Kbufc * (CaBuf + dCai + Cai);
    s0->Cai[i][j] = (sqrt(bc * bc + 4 * cc) - bc) / 2;

    double dNai = 0.0;
    double dKi = 0.0;
    if (state == 1)
    {
        dNai = -(INa + INaL + IbNa + 3 * INaK + 3 * INaCa) * inverseVcF * CAPACITANCE;
        dKi = -(Istim + IK1 + Ito + IKr + IKs + IKATP - 2 * INaK + IpK) * inverseVcF * CAPACITANCE;
    }
    else
    {
        dNai = -(INa + IbNa + 3 * INaK + 3 * INaCa) * inverseVcF * CAPACITANCE;
        dKi = -(Istim + IK1 + Ito + IKr + IKs - 2 * INaK + IpK) * inverseVcF * CAPACITANCE;
    }

    s0->Nai[i][j] += HT * dNai;
    s0->Ki[i][j] += HT * dKi;
    //compute steady state values and time constants
    double AM = 1. / (1. + exp((-60. - svolt) / 5.));
    double BM = 0.1 / (1. + exp((svolt + 35.) / 5.)) + 0.10 / (1. + exp((svolt - 50.) / 200.));
    double TAU_M = AM * BM;
    double M_INF = 1. / ((1. + exp((-55.5 - svolt) / 9.03)) * (1. + exp((-55.5 - svolt) / 9.03)));
    double AH_1 = 0;
    double AH_2 = 0;
    double AJ_1 = 0;
    double AJ_2 = 0;
    double BH_1 = 0;
    double BH_2 = 0;
    double BJ_1 = 0;
    double BJ_2 = 0;
    double TAU_H = 0;
    double TAU_J = 0;
    if (svolt >= -40.)
    {
        AH_1 = 0.;
        BH_1 = (0.77 / (0.13 * (1. + exp(-(svolt + 10.66) / 11.1))));
        TAU_H = 1.0 / (AH_1 + BH_1);
    }
    else
    {
        AH_2 = (0.057 * exp(-(svolt + 80.) / 6.8));
        BH_2 = (2.7 * exp(0.079 * svolt) + (3.1e5) * exp(0.3485 * svolt));
        TAU_H = 1.0 / (AH_2 + BH_2);
    }
    double H_INF = 1. / ((1. + exp((svolt + 71.55) / 7.43)) * (1. + exp((svolt + 71.55) / 7.43)));
    if (svolt >= -40.)
    {
        AJ_1 = 0.;
        BJ_1 = (0.6 * exp((0.057) * svolt) / (1. + exp(-0.1 * (svolt + 32.))));
        TAU_J = 1.0 / (AJ_1 + BJ_1);
    }
    else
    {
        AJ_2 = (((-2.5428e4) * exp(0.2444 * svolt) - (6.948e-6) *
                                                     exp(-0.04391 * svolt)) * (svolt + 37.78) /
                (1. + exp(0.311 * (svolt + 79.23))));
        BJ_2 = (0.02424 * exp(-0.01052 * svolt) / (1. + exp(-0.1378 * (svolt + 40.14))));
        TAU_J = 1.0 / (AJ_2 + BJ_2);
    }
    double J_INF = H_INF;
    double Xr1_INF = 1. / (1. + exp((-26. - svolt) / 7.));
    double axr1 = 450. / (1. + exp((-45. - svolt) / 10.));
    double bxr1 = 6. / (1. + exp((svolt - (-30.)) / 11.5));
    double TAU_Xr1 = axr1 * bxr1;
    double Xr2_INF = 1. / (1. + exp((svolt - (-88.)) / 24.));
    double axr2 = 3. / (1. + exp((-60. - svolt) / 20.));
    double bxr2 = 1.12 / (1. + exp((svolt - 60.) / 20.));
    double TAU_Xr2 = axr2 * bxr2;

    double Xs_INF = 1. / (1. + exp((-5. - svolt) / 14.));
    double Axs = (1400. / (sqrt(1. + exp((5. - svolt) / 6))));
    double Bxs = (1. / (1. + exp((svolt - 35.) / 15.)));
    double TAU_Xs = Axs * Bxs + 80;

    double R_INF,S_INF,TAU_R,TAU_S;
    if (location == EPI)
    {
        if (state == 1)
        {
            R_INF = 1. / (1. + exp((27.2 - svolt) / 6.));
            S_INF = 1. / (1. + exp((svolt + 6.3) / 5.));
        }
        else
        {
            R_INF = 1. / (1. + exp((20 - svolt) / 6.));
            S_INF = 1. / (1. + exp((svolt + 20) / 5.));
        }
        TAU_R = 9.5 * exp(-(svolt + 40.) * (svolt + 40.) / 1800.) + 0.8;
        TAU_S = 85. * exp(-(svolt + 45.) * (svolt + 45.) / 320.) + 5. / (1. + exp((svolt - 20.) / 5.)) + 3.;
    }
    else if (location == ENDO)
    {
        R_INF = 1. / (1. + exp((20 - svolt) / 6.));
        S_INF = 1. / (1. + exp((svolt + 28) / 5.));
        TAU_R = 9.5 * exp(-(svolt + 40.) * (svolt + 40.) / 1800.) + 0.8;
        TAU_S = 1000. * exp(-(svolt + 67) * (svolt + 67) / 1000.) + 8.;
    }
    else
    {
        R_INF = 1. / (1. + exp((20 - svolt) / 6.));
        S_INF = 1. / (1. + exp((svolt + 20) / 5.));
        TAU_R = 9.5 * exp(-(svolt + 40.) * (svolt + 40.) / 1800.) + 0.8;
        TAU_S = 85. * exp(-(svolt + 45.) * (svolt + 45.) / 320.) + 5. / (1. + exp((svolt - 20.) / 5.)) + 3.;
    }

    double D_INF = 1. / (1. + exp((-8 - svolt) / 7.5));
    double Ad = 1.4 / (1. + exp((-35 - svolt) / 13)) + 0.25;
    double Bd = 1.4 / (1. + exp((svolt + 5) / 5));
    double Cd = 1. / (1. + exp((50 - svolt) / 20));
    double TAU_D = Ad * Bd + Cd;
    double F_INF = 1. / (1. + exp((svolt + 20) / 7));
    double Af = 1102.5 * exp(-(svolt + 27) * (svolt + 27) / 225);
    double Bf = 200. / (1 + exp((13 - svolt) / 10.));
    double Cf = (180. / (1 + exp((svolt + 30) / 10))) + 20;
    double TAU_F = Af + Bf + Cf;
    double F2_INF = 0.67 / (1. + exp((svolt + 35) / 7)) + 0.33;
    double Af2 = 600 * exp(-(svolt + 25) * (svolt + 25) / 170);
    double Bf2 = 31 / (1. + exp((25 - svolt) / 10));
    double Cf2 = 16 / (1. + exp((svolt + 30) / 10));
    double TAU_F2 = Af2 + Bf2 + Cf2;
    double FCaSS_INF = 0.6 / (1 + (CaSS / 0.05) * (CaSS / 0.05)) + 0.4;
    double TAU_FCaSS = 80. / (1 + (CaSS / 0.05) * (CaSS / 0.05)) + 2.;
    //Update gates
    s0->M[i][j] = M_INF - (M_INF - sm) * exp(-HT / TAU_M);
    s0->H[i][j] = H_INF - (H_INF - sh) * exp(-HT / TAU_H);
    s0->J[i][j] = J_INF - (J_INF - sj) * exp(-HT / TAU_J);
    s0->Xr1[i][j] = Xr1_INF - (Xr1_INF - sxr1) * exp(-HT / TAU_Xr1);
    s0->Xr2[i][j] = Xr2_INF - (Xr2_INF - sxr2) * exp(-HT / TAU_Xr2);
    s0->Xs[i][j] = Xs_INF - (Xs_INF - sxs) * exp(-HT / TAU_Xs);
    s0->S[i][j] = S_INF - (S_INF - ss) * exp(-HT / TAU_S);
    s0->Rr[i][j] = R_INF - (R_INF - sr) * exp(-HT / TAU_R);
    s0->D[i][j] = D_INF - (D_INF - sd) * exp(-HT / TAU_D);
    s0->Ff[i][j] = F_INF - (F_INF - sf) * exp(-HT / TAU_F);
    s0->F2[i][j] = F2_INF - (F2_INF - sf2) * exp(-HT / TAU_F2);
    s0->FCass[i][j] = FCaSS_INF - (FCaSS_INF - sfcass) * exp(-HT / TAU_FCaSS);

    s0->dCai[i][j] = dCai;
    s0->IbCa[i][j] = IbCa;
    s0->IpCa[i][j] = IpCa;
    s0->INaCa[i][j] = INaCa;
    s0->Iup[i][j] = Iup;
    s0->Ileak[i][j] = Ileak;
    s0->Ixfer[i][j] = Ixfer;

}

void freeCanine(Atria_2D* c0)
{
    delete[] c0->V;
    delete[] c0->Cai;
    delete[] c0->CaSR;
    delete[] c0->CaSS;
    delete[] c0->Nai;
    delete[] c0->Ki;
    delete[] c0->M;
    delete[] c0->H;
    delete[] c0->J;
    delete[] c0->Xr1;
    delete[] c0->Xr2;
    delete[] c0->Xs;
    delete[] c0->Rr;
    delete[] c0->S;
    delete[] c0->D;
    delete[] c0->Ff;
    delete[] c0->F2;
    delete[] c0->FCass;
    delete[] c0->RR;
    delete[] c0->OO;
    delete[] c0->Itot;
    delete[] c0->du;
    delete[] c0->type;
    delete[] c0->location;
    delete[] c0->state;

    delete[] c0->dCai;
    delete[] c0->IbCa;
    delete[] c0->IpCa;
    delete[] c0->INaCa;
    delete[] c0->Iup;
    delete[] c0->Ileak;
    delete[] c0->Ixfer;
}

void cudaFreeCanine(Atria_2D* c0)
{
    hipFree(c0->V);
    hipFree(c0->Cai);
    hipFree(c0->CaSR);
    hipFree(c0->CaSS);
    hipFree(c0->Nai);
    hipFree(c0->Ki);
    hipFree(c0->M);
    hipFree(c0->H);
    hipFree(c0->J);
    hipFree(c0->Xr1);
    hipFree(c0->Xr2);
    hipFree(c0->Xs);
    hipFree(c0->Rr);
    hipFree(c0->S);
    hipFree(c0->D);
    hipFree(c0->Ff);
    hipFree(c0->F2);
    hipFree(c0->FCass);
    hipFree(c0->RR);
    hipFree(c0->OO);
    hipFree(c0->Itot);
    hipFree(c0->du);
    hipFree(c0->type);
    hipFree(c0->location);
    hipFree(c0->state);


    hipFree(c0->dCai);
    hipFree(c0->IbCa);
    hipFree(c0->IpCa);
    hipFree(c0->INaCa);
    hipFree(c0->Iup);
    hipFree(c0->Ileak);
    hipFree(c0->Ixfer);

}
